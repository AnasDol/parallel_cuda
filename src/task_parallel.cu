#include <stdio.h>
#include <math.h>
#include <string.h>
#include <stdlib.h>
#include <omp.h>
#include <hip/hip_runtime.h>


#define MAX_COUNT 10

__device__ void print_var(int* var, int size);
__device__ void print_matrix(int* matrix, int n);
 __device__ void print_triangle(int* matrix, int n, int* var, int size);

 __device__ int get_sum(int* matrix, int n, int* var, int size);

 __device__ void save_log(FILE* logfile, int** matrix, int n, int* var, int size, int sum, int min_sum);
__device__ void _variate(int* matrix, int n, int* var, int size, int min, int max, int deep, int* result_var, int* min_sum, FILE* logfile);

__device__ void new_variate(int* matrix, int n, int* var, int size, int min, int max, int true_max, int deep, int* result_var, int* min_sum, FILE* logfile, int* num);
 __global__ void variate(int* matrix, int n, int* var, int size, int min, int max, int deep, int* result_var, int* min_sum, FILE* logfile);
__global__ void startCompution(int* matrix, int n, int* var, int size, int* min_sum);
void host_print_var(int* var, int size);
void host_print_matrix(int* matrix, int n);
void host_print_triangle(int* matrix, int n, int* var, int size);

int host_get_sum(int* matrix, int n, int* var, int size);
//void new_variate(int* matrix, int n, int* var, int size, int min, int max, int deep, int* result_var, int* min_sum, FILE* logfile);
void new_variate(int* count, int* matrix, int n, int* var, int size, int min, int max, int deep, int* result_var, int* min_sum, FILE* logfile, int* device_var, int* device_result_var, int* device_matrix, int* device_min_sum);


int main(int argc, char* argv[]) {

    int n = 8;

    int* matrix = (int*)malloc(n * n * sizeof(int));

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            //fread(&matrix[i * n + j], sizeof(int), 1, file);
            matrix[i * n + j] = rand() % 10 + 1;
        }
    }

    if (n <= 20) host_print_matrix(matrix, n);

    int size = 3;

    int* var = (int*)malloc(size*sizeof(int));
    int* result_var = (int*)malloc(size*sizeof(int));
    int* device_var;
    hipMalloc((void**)&device_var, sizeof(int) * size);
    int* device_result_var;
    hipMalloc((void**)&device_result_var, sizeof(int) * size);

    int* device_matrix;
    hipMalloc((void**)&device_matrix, sizeof(int) * n * n);
    hipMemcpy(device_matrix, matrix, sizeof(int) * n * n, hipMemcpyHostToDevice);

    int min_sum = INT_MAX;
    int* device_min_sum;
    hipMalloc((void**)&device_min_sum, sizeof(int));
    hipMemcpy(device_min_sum, &min_sum, sizeof(int), hipMemcpyHostToDevice);

    




    int count = 0;

    new_variate(&count, matrix, n, var, size, 1, 8, 0, result_var, &min_sum, NULL, device_var, device_result_var, device_matrix, device_min_sum);











    // cudaMemcpy(result_var, device_result_var, sizeof(int) * size, cudaMemcpyDeviceToHost);
    // cudaMemcpy(&min_sum, device_min_sum, sizeof(int), cudaMemcpyDeviceToHost);

    //cudaDeviceSynchronize();

    // printf("\n---------------Output----------------\n");
    // printf("indexes:\n");
    // printf("[ ");
    // for (int i = 0; i < size; i++) {
    //     printf("%d ", result_var[i]);
    // }
    // printf("]");
    // printf("\n");
    // printf("submatrix:\n");
    // int row_index = 0, col_index = 0;
    // for (int i = result_var[row_index]; row_index < size; col_index = 0, row_index++, i = result_var[row_index]) {
    //     for (int j = result_var[col_index]; j <= i && col_index < size; col_index++, j = result_var[col_index]) {
    //         printf("%d ", matrix[(i - 1) * n + (j - 1)]);
    //     }
    //     printf("\n");
    // }
    // printf("min_sum: %d\n", min_sum);
    //printf("Time: %lf\n", finish - start);
    
    free(matrix);
    free(var);
    free(result_var);

    hipFree(device_matrix);
    hipFree(device_var);
    hipFree(device_result_var);
    
    return 0;
}

__device__ void print_var(int* var, int size) {
    printf("[ ");
    for (int i = 0; i < size; i++) {
        printf("%d ", var[i]);
    }
    printf("]");
    printf("\n");
}

void host_print_var(int* var, int size) {
    printf("[ ");
    for (int i = 0; i < size; i++) {
        printf("%d ", var[i]);
    }
    printf("]");
    printf("\n");
}

__device__ void print_matrix(int* matrix, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("%d ", matrix[i * n + j]);
        }
        printf("\n");
    }

}

void host_print_matrix(int* matrix, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("%d ", matrix[i * n + j]);
        }
        printf("\n");
    }

}

 __device__ void print_triangle(int* matrix, int n, int* var, int size) {

    int row_index = 0, col_index = 0;
    for (int i = var[row_index]; row_index < size; col_index = 0, row_index++, i = var[row_index]) {
        for (int j = var[col_index]; j <= i && col_index < size; col_index++, j = var[col_index]) {
            printf("%d ", matrix[(i - 1) * n + (j - 1)]);
        }
        printf("\n");
    }
}

void host_print_triangle(int* matrix, int n, int* var, int size) {

    int row_index = 0, col_index = 0;
    for (int i = var[row_index]; row_index < size; col_index = 0, row_index++, i = var[row_index]) {
        for (int j = var[col_index]; j <= i && col_index < size; col_index++, j = var[col_index]) {
            printf("%d ", matrix[(i - 1) * n + (j - 1)]);
        }
        printf("\n");
    }
}

 __device__ int get_sum(int* matrix, int n, int* var, int size) {

    int sum = 0;

    int row_index = 0, col_index = 0;
    for (int i = var[row_index]; row_index < size; col_index = 0, row_index++, i = var[row_index]) {
        for (int j = var[col_index]; j <= i && col_index < size; col_index++, j = var[col_index]) {
            sum += matrix[(i - 1) * n + (j - 1)];
        }
    }
    return sum;

}

 int host_get_sum(int* matrix, int n, int* var, int size) {

    int sum = 0;

    int row_index = 0, col_index = 0;
    for (int i = var[row_index]; row_index < size; col_index = 0, row_index++, i = var[row_index]) {
        for (int j = var[col_index]; j <= i && col_index < size; col_index++, j = var[col_index]) {
            sum += matrix[(i - 1) * n + (j - 1)];
        }
    }
    return sum;

}

__global__ void startCompution(int* matrix, int n, int* var, int size, int* min_sum) {

    int threadId = threadIdx.x;  // Идентификатор потока внутри блока
    int blockId = blockIdx.x;    // Идентификатор блока внутри сетки
    printf("Thread ID: %d, Block ID: %d\n", threadId, blockId);

}

void new_variate(int* count, int* matrix, int n, int* var, int size, int min, int max, int deep, int* result_var, int* min_sum, FILE* logfile, int* device_var, int* device_result_var, int* device_matrix, int* device_min_sum) {

    for (int i = min; i <= max - size + 1; i++) {
        if (deep < size) {
            var[deep] = i;
            new_variate(count, matrix, n, var, size, i + 1, max + 1, deep + 1, result_var, min_sum, logfile, device_var, device_result_var, device_matrix, device_min_sum);
        }
        else {

            (*count)++;

            if (*count >= MAX_COUNT) {
                
                hipDeviceSynchronize();
                hipMemcpy(var, device_var, sizeof(int) * size, hipMemcpyHostToDevice);

                startCompution<<<1,1>>>(matrix, n, device_var, size, device_min_sum);
                *count = 0;

            }

            host_print_var(var, size);
            
            int sum = host_get_sum(matrix, n, var, size);
            if (sum < *min_sum) {
                *min_sum = sum;
                for (int j = 0; j < size; j++) result_var[j] = var[j];
            }

            break;
        }
    }

    if (deep == 0) {
        hipDeviceSynchronize();
        hipMemcpy(var, device_var, sizeof(int) * size, hipMemcpyHostToDevice);

        startCompution<<<1,1>>>(matrix, n, device_var, size, device_min_sum);
        *count = 0;
    }

}




